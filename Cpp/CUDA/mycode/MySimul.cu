#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>

#include <iostream>

__global__ void init_zero(double *d_x){
  d_x[blockIdx.x] = 0.0;
}

__global__ void randomWalk(double *d_x, double *d_GPU){
  d_x[blockIdx.x] = d_x[blockIdx.x] + 2.0*d_GPU[blockIdx.x]-1.0;
}

int main() {
  using namespace std;
  hiprandGenerator_t genGPU;

  hiprandCreateGenerator(&genGPU, HIPRAND_RNG_PSEUDO_MTGP32);
  hiprandSetPseudoRandomGeneratorSeed(genGPU, 1234ULL);

  const int n = 10000;
  double GPU[n];
  double x[n];

  // Allocated memory of the GPU
  double* d_GPU;
  double* d_x;
  hipMalloc(&d_GPU, n*sizeof(double));
  hipMalloc(&d_x,   n*sizeof(double));

  init_zero<<<n, 1>>>(d_x);

  for ( int i=0; i<100000; i++){
    // Generate an array of random numbers in the GPU
    hiprandGenerateUniformDouble(genGPU, d_GPU, n);
    randomWalk <<< n,1 >>> (d_x, d_GPU);
  }

  // Copy from GPU to CPU
  // cudaMemcpy(GPU, d_GPU, n*sizeof(double), cudaMemcpyDeviceToHost);
  // cudaMemcpy(x,   d_x,   n*sizeof(double), cudaMemcpyDeviceToHost);
  // for ( int i = 0; i < n; ++i ) {
  //   cout << GPU[i] << ' ' << x[i] << endl;
  // }

  hiprandDestroyGenerator(genGPU);
  hipFree(d_GPU);
  hipFree(d_x);

}
